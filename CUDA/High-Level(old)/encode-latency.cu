// C++ IMPORTS 
#include <iostream>
#include <bitset>
#include <random>

// CUSTOM IMPORTS
#include "encode_library.h"

using namespace std;

int main()
{

	uint256_t* piece, * d_piece;
	hipHostMalloc(&piece, 4 * sizeof(unsigned long long) * 128);

	unsigned char* byte_piece = (unsigned char*)piece;
	for (int i = 0; i < 32 * 128; i++)
	{
		byte_piece[i] = 5u;
	}

	uint256_t expanded_iv;

	unsigned char* byte_expanded_iv = (unsigned char*)&expanded_iv;
	for (int i = 0; i < 32; i++)
	{
		byte_expanded_iv[i] = 3u;
	}

	hipMalloc(&d_piece, 4 * sizeof(unsigned long long) * 128);
	hipMemcpyAsync(d_piece, piece, 4 * sizeof(unsigned long long) * 128, hipMemcpyHostToDevice, 0);

	encode_test_new << <1, 1 >> > (d_piece, expanded_iv);

	hipMemcpyAsync(d_piece, piece, 4 * sizeof(unsigned long long) * 128, hipMemcpyHostToDevice, 0);

	encode_test_mont << <1, 1 >> > (d_piece, expanded_iv);

	hipMemcpyAsync(d_piece, piece, 4 * sizeof(unsigned long long) * 128, hipMemcpyHostToDevice, 0);

	encode_test_old << <1, 1 >> > (d_piece, expanded_iv);

	hipMemcpyAsync(piece, d_piece, 4 * sizeof(unsigned long long) * 128, hipMemcpyDeviceToHost, 0);
	hipDeviceSynchronize();

	/*unsigned char* piece_byte_ptr = (unsigned char*)piece;
	for (int i = 0; i < 128 * 32; i++)
	{
		unsigned number = (unsigned)piece_byte_ptr[i];

		if (number == 0)
		{
			cout << "00";
		}
		else if (number < 16)
		{
			cout << "0";
			cout << hex << number;
		}
		else
		{
			cout << hex << number;
		}

		if (i % 32 == 31)
			cout << endl;
	}
	*/
	return 0;
}