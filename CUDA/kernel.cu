#include "hip/hip_runtime.h"
// C++ IMPORTS 
#include <iostream>
#include <bitset>

// CUSTOM IMPORTS
#include "uint128.h"

using namespace std;

__global__ void device_mul(unsigned long long* a, unsigned long long* b, uint128_t* dres)
{
	*dres = mul64x2(*a, *b);
}

__global__ void device_equals(uint128_t* h1, uint128_t* h2, uint128_t * h3)
{   
	/* SETUP
	h1 != h2, 
    h2 = h3
	*/
	if (*h1 == *h2) {
		printf("Fail for h1==h2\n");
	}
	else {
		printf("Success for h1==h2\n");
	}
	if (*h2 == *h3) {
		printf("Success for h2==h3\n");
	}
	else {
		printf("Fail for h2==h3\n");
	}

}

__global__ void device_greater(uint128_t* h1, uint128_t* h2, uint128_t * h3)
{
	/* SETUP
	h1 < h2,
	h2 = h3
	*/
	if (*h1 > *h2) {
		printf("Fail for h1 > h2\n");
	}
	else {
		printf("Success for h1 > h2\n");
	}
	if (*h2 > *h3) {
		printf("Fail for h2 > h3\n");
	}
	else {
		printf("Success for h2 > h3\n");
	}

}

__global__ void device_lesser(uint128_t* h1, uint128_t* h2, uint128_t * h3)
{
	/* SETUP
	h1 < h2,
	h2 = h3
	*/
	if (*h1 < *h2) {
		printf("Success for h1 < h2\n");
	}
	else {
		printf("Fail for h1 < h2\n");
	}
	if (*h2 < *h3) {
		printf("Fail for h2 < h3\n");
	}
	else {
		printf("Success for h2 < h3\n");
	}

}


int main()
{

	// Multiplication test start

	unsigned long long a, b;

	a = 1289312831239555555;
	b = 1290390120391092390;

	uint128_t res;

	unsigned long long *da, *db;
	uint128_t *dres;
	hipMalloc(&da, sizeof(unsigned long long));
	hipMalloc(&db, sizeof(unsigned long long));
	hipMalloc(&dres, 2 * sizeof(unsigned long long));

	hipMemcpy(da, &a, sizeof(unsigned long long), hipMemcpyHostToDevice);
	hipMemcpy(db, &b, sizeof(unsigned long long), hipMemcpyHostToDevice);

	device_mul << <1, 1 >> > (da, db, dres);

	hipMemcpy(&res, dres, 2 * sizeof(unsigned long long), hipMemcpyDeviceToHost);

	cout << "Multiplication test result:\n" << bitset<64>(res.high) << endl << bitset<64>(res.low) << endl;

	// Multiplication test end

	/////////////////////////

	// `==` test start

	uint128_t h1, h2, h3;
	uint128_t *d1, *d2, *d3;

	h1.high = std::bitset<64>(std::string("1111111111111111111111111111111111111111111111111111111111111111")).to_ullong();
	h1.low = std::bitset<64>(std::string("1111111111111111111111110011111111111111001111100111111111111000")).to_ullong();

	h2.high = std::bitset<64>(std::string("1111111111111111111111111111111111111111111111111111111111111111")).to_ullong();
	h2.low = std::bitset<64>(std::string("1111111111111111111111110011111111111111001111100111111111111100")).to_ullong();

	h3.high = std::bitset<64>(std::string("1111111111111111111111111111111111111111111111111111111111111111")).to_ullong();
	h3.low = std::bitset<64>(std::string("1111111111111111111111110011111111111111001111100111111111111100")).to_ullong();

	hipMalloc(&d1, 2 * sizeof(unsigned long long));
	hipMalloc(&d2, 2 * sizeof(unsigned long long));
	hipMalloc(&d3, 2 * sizeof(unsigned long long));

	hipMemcpy(d1, &h1, 2 * sizeof(unsigned long long), hipMemcpyHostToDevice);
	hipMemcpy(d2, &h2, 2 * sizeof(unsigned long long), hipMemcpyHostToDevice);
	hipMemcpy(d3, &h3, 2 * sizeof(unsigned long long), hipMemcpyHostToDevice);

	device_equals << <1, 1 >> > (d1, d2, d3);

	// `==` test end

	/////////////////////////

	// `>` test start

	device_greater << <1, 1 >> > (d1, d2, d3);

	// `>` test end

	/////////////////////////

	// `<` test start

	device_lesser << <1, 1 >> > (d1, d2, d3);

	// `<` test end

	return 0;
}