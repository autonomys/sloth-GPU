#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <bitset>
#include <string>

#include "encode_ptx.h"

using namespace std;


#define CUDA_FATAL(expr) do {				\
    hipError_t code = expr;				\
    if (code != hipSuccess) {				\
        cerr << #expr << "@" << __LINE__ << " failed: "	\
             << hipGetErrorString(code) << endl;	\
	exit(1);					\
    }							\
} while(0)



int main()
{
// creating problem with nsight, can remove this part - BEGIN
    hipDeviceProp_t prop;
    CUDA_FATAL(hipGetDeviceProperties(&prop, 0));
    cout << prop.name << endl;
    cout << "Capability: " << prop.major << "." << prop.minor << endl;
    cout << "Clock rate: " << prop.clockRate << "kHz" << endl;
    cout << "Memory clock rate: " << prop.memoryClockRate << "kHz" << endl;
    cout << "L2 cache size: " << prop.l2CacheSize << endl;
    cout << "Shared Memory: " << prop.sharedMemPerBlock << endl;
    // creating problem with nsight, can remove this part - END


    // instead of below, we can give any number to blockSize and minGridSize like this:
    /*
    int blockSize = 256;
    int minGridSize = 30;
    */

    int blockSize;      // The launch configurator returned block size
    int minGridSize;    // The minimum grid size needed to achieve the
                        // maximum occupancy for a full device launch
    CUDA_FATAL(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                                  encode_ptx_test));  // creating problem with nsight, can remove this part also

    cout << "kernel<<<" << minGridSize << ", " << blockSize << ">>>, \n";  // shows the parameters for max-occupancy

	u32* piece = (u32*)malloc(sizeof(u32) * 8 * 128 * minGridSize * blockSize);  // allocates memory on the CPU for the piece
	u32* d_piece_ptx, * d_expanded_iv_ptx;  // creating device pointers

    hipMalloc(&d_piece_ptx, sizeof(u32) * 8 * 128 * minGridSize * blockSize);  // allocates memory on the GPU for the piece
    hipMalloc(&d_expanded_iv_ptx, sizeof(u32) * 8 * minGridSize * blockSize);  // allocates memory on the GPU for the expanded_iv
	// since expanded_iv will be static for a farmer, this does not need to be copied from CPU everytime, it can be hardcoded to GPU

    hipMemset(d_piece_ptx, 5u, sizeof(u32) * 8 * 128 * minGridSize * blockSize);  // setting all values inside piece as 5
    hipMemset(d_expanded_iv_ptx, 3u, sizeof(u32) * 8 * minGridSize * blockSize);  // setting all values inside expanded_iv as 3

    encode_ptx_test<<<minGridSize, blockSize >>>(d_piece_ptx, d_expanded_iv_ptx);  // calling the kernel

    hipMemcpy(piece, d_piece_ptx, sizeof(u32) * 8 * 128 * minGridSize * blockSize, hipMemcpyDeviceToHost);  // copying the result back to CPU

	hipDeviceSynchronize();  // wait for GPU operations to finish

    cout << "Operation successful!\n";

	// FOR DEBUGGING THE OUTPUT (prints the piece in hexadecimal)
	/*unsigned char* piece_byte_ptr = (unsigned char*)piece;
	for (int i = 0; i < 128 * 32; i++)
	{
		unsigned number = (unsigned)piece_byte_ptr[i];

		if (number == 0)
		{
			cout << "00";
		}
		else if (number < 16)
		{
			cout << "0";
			cout << hex << number;
		}
		else
		{
			cout << hex << number;
		}

		if (i % 32 == 31)
			cout << endl;
	}*/

    return 0;
}
