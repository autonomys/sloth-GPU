#include "hip/hip_runtime.h"
// C++ IMPORTS 
#include <iostream>
#include <bitset>

// CUSTOM IMPORTS
#include "sqrt_library.h"

using namespace std;

int main()
{
	uint256_t p, *d_p, a, *d_a, exp, *d_exp;
	uint512_t mu, *d_mu;
	unsigned k = 256;

	p.high.high = std::bitset<64>(std::string("1111111111111111111111111111111111111111111111111111111111111111")).to_ullong();
	p.high.low = std::bitset<64>(std::string("1111111111111111111111111111111111111111111111111111111111111111")).to_ullong();
	p.low.high = std::bitset<64>(std::string("1111111111111111111111111111111111111111111111111111111111111111")).to_ullong();
	p.low.low = std::bitset<64>(std::string("1111111111111111111111111111111111111111111111111111111101000011")).to_ullong();

	mu.low.low.low = std::bitset<64>(std::string("0000000000000000000000000000000000000000000000000000000010111101")).to_ullong();
	mu.high.low.low = 1ull;

	a.high.high = std::bitset<64>(std::string("1111111111111111111111111111111111111111111111111111111111111111")).to_ullong();
	a.high.low = std::bitset<64>(std::string("1111111111111111111111111111111111111111111111111111111111111111")).to_ullong();
	a.low.high = std::bitset<64>(std::string("1111111111111111111111111111111111111111111111111111111111111111")).to_ullong();
	a.low.low = std::bitset<64>(std::string("1111111111111111111111111111111111111111111111111111111100111100")).to_ullong();

	exp.high.high = std::bitset<64>(std::string("0011111111111111111111111111111111111111111111111111111111111111")).to_ullong();
	exp.high.low = std::bitset<64>(std::string("1111111111111111111111111111111111111111111111111111111111111111")).to_ullong();
	exp.low.high = std::bitset<64>(std::string("1111111111111111111111111111111111111111111111111111111111111111")).to_ullong();
	exp.low.low = std::bitset<64>(std::string("1111111111111111111111111111111111111111111111111111111111010001")).to_ullong();


	hipMalloc(&d_p, 4 * (sizeof(unsigned long long)));
	hipMalloc(&d_mu, 8 * (sizeof(unsigned long long)));
	hipMalloc(&d_a, 4 * (sizeof(unsigned long long)));
	hipMalloc(&d_exp, 4 * (sizeof(unsigned long long)));

	hipMemcpy(d_p, &p, 4 * (sizeof(unsigned long long)), hipMemcpyHostToDevice);
	hipMemcpy(d_mu, &mu, 8 * (sizeof(unsigned long long)), hipMemcpyHostToDevice);
	hipMemcpy(d_a, &a, 4 * (sizeof(unsigned long long)), hipMemcpyHostToDevice);
	hipMemcpy(d_exp, &exp, 4 * (sizeof(unsigned long long)), hipMemcpyHostToDevice);

	//cout << "a: " << bitset<64>(a.high.high) << bitset<64>(a.high.low) << bitset<64>(a.low.high) << bitset<64>(a.low.low) << endl;
	//cout << "p: " << bitset<64>(p.high.high) << bitset<64>(p.high.low) << bitset<64>(p.low.high) << bitset<64>(p.low.low) << endl;

	cout << "Square-root permutation test ...";
	sqrt_permutation << <1, 1 >> > (d_a, exp, p, mu, k);
	hipMemcpy(&a, d_a, 4 * sizeof(unsigned long long), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	if (bitset<64>(a.high.high) == std::bitset<64>(std::string("0101001110000101000111011100000110110011100000101100011000001000")) &&
		bitset<64>(a.high.low) == std::bitset<64>(std::string("0010000010111000010011001110101001110011111011111001000100111100")) &&
		bitset<64>(a.low.high) == std::bitset<64>(std::string("0001010000111011111111001001010100011110001110011010001110101101")) &&
		bitset<64>(a.low.low) == std::bitset<64>(std::string("1101011000011000111100111100100011101011110011110101100110111000"))
		) {

		printf("passed!\n");
	}
	else
	{
		printf("failed!\n");
	}

	//cout << bitset<64>(a.high.high) << endl << bitset<64>(a.high.low) << endl << bitset<64>(a.low.high) << endl << bitset<64>(a.low.low) << endl;

	return 0;
}